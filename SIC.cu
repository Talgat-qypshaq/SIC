#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
//#include "SIC_QPSK.c"
//#include "SIC_QAM16.c"
//#include "SIC_QAM64.c"
//#include "SIC_Rayleigh.c"
#include "SIC_header.cuh"
using namespace std;

FILE *fp1, *fp2, *fp3;
int lengthOfLineFunction(FILE *fp2, char *fileLocation, char *line, int lengthOfLine);

__device__ double QPSK(double signal)
{
	//printf("B signal: %.5f\n", signal);
	if (signal>0)
		signal = 1;
	else
		signal = -1;
	//printf("A signal: %.5f\n", signal);
	return signal;
}

__device__ double QAM16(double signal)
{
	//printf("B signal: %.5f\n", signal);
	if (signal < 0.25)
		signal = 1;
	if (signal >= 0.25 && signal < 0.5)
		signal = -1;
	if (signal >= 0.5 && signal < 0.75)
		signal = 3;
	if (signal >= 0.75)
		signal = -3;
	//printf("A signal: %.5f\n", signal);
	return signal;
}

__device__ double QAM64(double signal)
{
	//printf("B signal: %.5f\n", signal);
	if (signal < 0.03125)
		signal = 7;
	if (signal >= 0.03125 && signal < 0.0625)
		signal = 5;
	if (signal >= 0.0625 && signal < 0.75)
		signal = 3;
	if (signal >= 0.75 && signal < 0.09375)
		signal = 1;
	if (signal >= 0.09375 && signal < 0.125)
		signal = -1;
	if (signal >= 0.125 && signal < 0.15625)
		signal = -3;
	if (signal >= 0.15625 && signal < 0.1875)
		signal = -5;
	if (signal >= 0.1875)
		signal = -7;
	//printf("A signal: %.5f\n", signal);
	return signal;
}

__global__ void SIC(float *powerCoefficients, double *Rayleigh, double *receivedSignal)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	//int index = threadIdx.x;
	double signal[2];
	double sumSignalPowerCoefficientChannel[2];
	//index of the thread is the id of user, which is being decoded
	if (index < cellSize)
	{
		//printf("luck I am");
		//int order = index % numberOfUEs;
		int order = 1;
		for (int i = numberOfUEs; i >= order; i--)
		{
			//int order = index % i;
			//printf("oder = %d \n", order);
			signal[0] = (receivedSignal[index] - sumSignalPowerCoefficientChannel[0]) / (Rayleigh[index]*powerCoefficients[i]);
			signal[1] = (receivedSignal[index + cellSize] - sumSignalPowerCoefficientChannel[1]) / (Rayleigh[index + cellSize] * powerCoefficients[i]);
			switch (modulation)
			{
				case 4:
				{
					 signal[0] = QPSK(signal[0]);
					 signal[1] = QPSK(signal[1]);
					 break;
				}
				case 16:
				{
					 signal[0] = QAM16(signal[0]);
					 signal[1] = QAM16(signal[1]);
					 break;
				}
				case 64:
				{
					 signal[0] = QAM64(signal[0]);
					 signal[1] = QAM64(signal[1]);
					 break;
				}
			}
			if (i != order)
			{
				sumSignalPowerCoefficientChannel[0] = sumSignalPowerCoefficientChannel[0] + (Rayleigh[index] * signal[0] * powerCoefficients[i]);
				sumSignalPowerCoefficientChannel[1] = sumSignalPowerCoefficientChannel[1] + (Rayleigh[index + cellSize] * signal[1] * powerCoefficients[i]);
			}
		}
		//printf("RayleighReal = %.3f; receivedSignalReal = %.3f;\n", Rayleigh[index], receivedSignal[index]);
		//printf("RayleighImag = %.3f; receivedSignalImag = %.3f;\n", Rayleigh[index + numberOfUEs], receivedSignal[index + numberOfUEs]);
		receivedSignal[index] = signal[0];
		receivedSignal[index + cellSize] = signal[1];
	}
}

int main(void )
{
	char fileLocation[256] = "/home/talgat/github/SIC/OPA/PowerAllocation";
	int totalPower = 1;
	int order = cellSize % numberOfUEs;
	char a[8] = "10";
	char fileLocationEnd[8] = ".txt";
	strcat(a, fileLocationEnd);
	strcat(fileLocation, a);
	//printf("File Location %s \n", fileLocation);
	char line[255];
	int lengthOfLine = 0;
	int i = 0;
	lengthOfLine = lengthOfLineFunction(fp1, fileLocation, line, lengthOfLine);
	//printf("\nlengthOfLine %d \n", lengthOfLine);
	//array for optimum power allocation coefficients
	float *powerCoefficientMatrix = 0;
	if (powerCoefficientMatrix != 0)
	{
		powerCoefficientMatrix = (float*)realloc(powerCoefficientMatrix, numberOfUEs * sizeof(float));
	}
	else
	{
		powerCoefficientMatrix = (float*)malloc(numberOfUEs * sizeof(float));
	}
	//iterate through each value in a line
	fp2 = fopen(fileLocation, "r");
	fgets(line, lengthOfLine, (FILE*)fp2);
	//printf("\nline: %s\n", line);
	char *p = strtok(line, " ");
	for (int m = 0; m < numberOfUEs; m++)
	{
		powerCoefficientMatrix[m] = (float)atof(p);
		p = strtok(NULL, " ");
	}
	fclose(fp2);

		int *generatedSignal;
		switch (modulation)
		{
			case 4:
			{
				generatedSignal = getGeneratedQPSKSignal();
				//printf("4\n");
				break;
			}
			case 16:
			{
				generatedSignal = getGeneratedQAM16Signal();
				//printf("16\n");
				break;
			}
			case 64:
			{
				generatedSignal = getGeneratedQAM64Signal();
				//printf("64\n");
				break;
			}
		}

		double *rayleighChannel;
		rayleighChannel = getGeneratedRayleighChannel(powerCoefficientMatrix);
		float signalWithPowerCoefficient[cellSize * 2];

		for (i = 0; i<cellSize; i++)
		{
			signalWithPowerCoefficient[i] = generatedSignal[i] * sqrt(powerCoefficientMatrix[order] * totalPower);
			signalWithPowerCoefficient[i + cellSize] = generatedSignal[i + cellSize] * sqrt(powerCoefficientMatrix[order] * totalPower);
		}

		float superSignalReal = 0;
		float superSignalImag = 0;

		for (i = 0; i < cellSize; i++)
		{
			superSignalReal = superSignalReal + signalWithPowerCoefficient[i];
			superSignalImag = superSignalImag + signalWithPowerCoefficient[i + cellSize];
		}

		double receivedSignal[cellSize * 2];
		for (i = 0; i < cellSize; i++)
		{
			//noise is considered as zero
			receivedSignal[i] = superSignalReal*rayleighChannel[i];
			receivedSignal[i + cellSize] = superSignalImag*rayleighChannel[i + cellSize];
		}

		float  *dev_PowerCoefficientMatrix;
		double  *dev_RayleighChannel;
		double  *dev_ReceivedSignal;

		hipMalloc((void**)&dev_PowerCoefficientMatrix, numberOfUEs * sizeof(float));
		hipMemcpy(dev_PowerCoefficientMatrix, powerCoefficientMatrix, numberOfUEs * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_RayleighChannel, cellSize * 2 * sizeof(double));
		hipMemcpy(dev_RayleighChannel, rayleighChannel, cellSize * 2 * sizeof(double), hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_ReceivedSignal, cellSize * 2 * sizeof(double));
		hipMemcpy(dev_ReceivedSignal, receivedSignal, cellSize * 2 * sizeof(double), hipMemcpyHostToDevice);

		hipEvent_t start, stop;
		float elapsedTime;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		//CALLING CUDA START  ****************************************************************************************
		SIC << <cellCoefficient, groupSize >> > (dev_PowerCoefficientMatrix, dev_RayleighChannel, dev_ReceivedSignal);
		//CALLING CUDA END    ****************************************************************************************
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);
		
		char fileLocation_2[128] = "/home/talgat/github/SIC/results.txt";
		fp3 = fopen(fileLocation_2, "w");
		printf("Time to generate: %.3f ms", elapsedTime);
		fprintf(fp3, "%.3f", elapsedTime);
		fclose(fp3);
		printf("\n");
		hipMemcpy(&receivedSignal, dev_ReceivedSignal, cellSize * 2 * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(dev_PowerCoefficientMatrix);
		hipFree(dev_RayleighChannel);
		hipFree(dev_ReceivedSignal);
}
int lengthOfLineFunction(FILE *fp, char *fileLocation, char *line, int lengthOfLine)
{
	fp = fopen(fileLocation, "r");
	int numberOfUsers = 1;
	for (int i = 0; i < 1; i++) {
		fgets(line, 511, (FILE*)fp);
		lengthOfLine = strlen(line);
		for (int j = 0; line[j] != '\0'; j++) {
			if (line[j] == ' ') numberOfUsers++;
		}
	}
	//printf("1. There are %d chars in a line \n", lengthOfLine);
	//printf("2. There are %d UEs in a BS", numberOfUsers);
	lengthOfLine = lengthOfLine + 3;
	fclose(fp);
	return lengthOfLine;
}
